#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>


#define N 500000
#define K 5
#define MAX_ITER 20
#define TPB 128
#define EPSILON 0.00001


__device__ float Euclidean_Distance(const float x1, const float x2, const float y1, const float y2)
{
    return sqrt(pow((x1-y1),2) + pow((x2-y2),2));
}


__global__ void Cluster_Assignment(const float *d_Point_Coord_x, const float *d_Point_Coord_y, int *d_Cluster_Membership, const float *d_Centroid_Coord_X, const float *d_Centroid_Coord_Y)
{
    //get idx for this datapoint
    const int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= N) return;


    //find the closest centroid to this datapoint
    float min_dist = INFINITY;
    int closest_centroid = 0;

    for(int c = 0; c<K; ++c)

    {
        float dist = Euclidean_Distance(d_Point_Coord_x[idx], d_Point_Coord_y[idx], d_Centroid_Coord_X[c], d_Centroid_Coord_Y[c]);

        if(dist < min_dist)
        {
            min_dist = dist;
            closest_centroid=c;
        }
    }

    //assign closest cluster id for this datapoint/thread
    d_Cluster_Membership[idx]=closest_centroid;
}
__global__ void Sum_Update(const float *d_Point_Coord_x, const float *d_Point_Coord_y, const int *d_Cluster_Membership, float *d_Centroid_sum_x, float *d_Centroid_sum_y, int *d_Cluster_Size) {

    //get idx of thread at grid level
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N) return;

    int clust_id = d_Cluster_Membership[idx];

    atomicAdd(&(d_Centroid_sum_x[clust_id]), d_Point_Coord_x[idx]);
    atomicAdd(&(d_Centroid_sum_y[clust_id]), d_Point_Coord_y[idx]);
    atomicAdd(&(d_Cluster_Size[clust_id]), 1);


}

__host__ void read_data(const char *file_name, float *h_Point_Coord_x, float *h_Point_Coord_y) {
    FILE *file;
    file = fopen(file_name, "r");
    //printf("%s\n", file_name);
    //initalize datapoints from csv
    printf("DataPoints: \n");
    for(int i=0;i<N;++i){
        fscanf(file,"%f,%f\n", &h_Point_Coord_x[i], &h_Point_Coord_y[i]);
        //printf("(%f, %f) \n",  h_Point_Coord_x[i], h_Point_Coord_y[i]);
    }
    fclose(file);
}
                 
__host__ void initalize_cluster(float *h_Point_Coord_x,float *h_Point_Coord_y,float *h_Centroid_Coord_X,float *h_Centroid_Coord_Y, float *h_Centroid_sum_x, float *h_Centroid_sum_y, int *h_Cluster_Size){
    
    //printf("Clusters: \n");
    for(int i=0;i<K;++i){
        int r = rand() % N;
        h_Centroid_sum_x[i]=0.0;
        h_Centroid_sum_y[i]=0.0;
        h_Cluster_Size[i]=0;
        h_Centroid_Coord_X[i] = h_Point_Coord_x[r];
        h_Centroid_Coord_Y[i] = h_Point_Coord_y[r];
        
    }
}
int main()
{
    
    srand(time(NULL));   
    
    const char *file_name = "/content/drive/My Drive/Parallel/K-means/Cuda/2D_data_1000.csv";

    //allocate memory on the device for the data points
    float *d_Point_Coord_x;
    float *d_Point_Coord_y;
    //allocate memory on the device for the cluster assignments
    int *d_Cluster_Membership;
    //allocate memory on the device for the cluster centroids
    float *d_Centroid_sum_x;
    float *d_Centroid_sum_y;
    float *d_Centroid_Coord_X;
    float *d_Centroid_Coord_Y;
    //allocate memory on the device for the cluster sizes
    int *d_Cluster_Size;

    hipMalloc(&d_Point_Coord_x, N*sizeof(float));
    hipMalloc(&d_Point_Coord_y, N*sizeof(float));
    hipMalloc(&d_Cluster_Membership,N*sizeof(int));
    hipMalloc(&d_Centroid_sum_x,K*sizeof(float));
    hipMalloc(&d_Centroid_sum_y,K*sizeof(float));
    hipMalloc(&d_Centroid_Coord_X,K*sizeof(float));
    hipMalloc(&d_Centroid_Coord_Y,K*sizeof(float));
    hipMalloc(&d_Cluster_Size,K*sizeof(int));

    //allocate memory for host
    float *h_Centroid_Coord_X = (float*)malloc(K*sizeof(float));
    float *h_Centroid_Coord_Y = (float*)malloc(K*sizeof(float));
    float *h_Centroid_sum_x = (float*)malloc(K*sizeof(float));
    float *h_Centroid_sum_y = (float*)malloc(K*sizeof(float));
    float *h_Point_Coord_x = (float*)malloc(N*sizeof(float));
    float *h_Point_Coord_y = (float*)malloc(N*sizeof(float));
    int *h_Cluster_Membership = (int*)malloc(N*sizeof(int));
    int *h_Cluster_Size = (int*)malloc(K*sizeof(int));


    //Read data from CSV
    read_data(file_name,h_Point_Coord_x,h_Point_Coord_y);
    //Initialization of cluster from datapoints
    initalize_cluster(h_Centroid_Coord_X,h_Centroid_Coord_Y,h_Centroid_Coord_X,h_Centroid_Coord_Y, h_Centroid_sum_x, h_Centroid_sum_y, h_Cluster_Size);



    //copy datapoints and all other data from host to device
    hipMemcpy(d_Centroid_Coord_X,h_Centroid_Coord_X,K*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_Centroid_Coord_Y,h_Centroid_Coord_Y,K*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_Centroid_sum_x,h_Centroid_sum_x,K*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_Centroid_sum_y,h_Centroid_sum_y,K*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_Point_Coord_x,h_Point_Coord_x,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_Point_Coord_y,h_Point_Coord_y,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_Cluster_Size,h_Cluster_Size,K*sizeof(int),hipMemcpyHostToDevice);



    //Start time for clustering
    clock_t start = clock();
    int cur_iter = 0;
    while(cur_iter < MAX_ITER)
    {
        printf("Iter %d: \n",cur_iter);
        //Start time for iteration
        clock_t start_iter = clock();


        //Points assg
        Cluster_Assignment<<<(N+TPB-1)/TPB, TPB>>>(d_Point_Coord_x, d_Point_Coord_y, d_Cluster_Membership, d_Centroid_Coord_X, d_Centroid_Coord_Y);


        //reset centroids and cluster sizes (will be updated in the next kernel)
        hipMemset(d_Centroid_sum_x,0.0,K*sizeof(float));
        hipMemset(d_Centroid_sum_y,0.0,K*sizeof(float));

        //call centroid update
        Sum_Update<<<(N+TPB-1)/TPB, TPB>>>(d_Point_Coord_x, d_Point_Coord_y, d_Cluster_Membership, d_Centroid_sum_x, d_Centroid_sum_y, d_Cluster_Size);

        //Copy centroids sum and clusters sizes back to host
        hipMemcpy(h_Centroid_sum_x,d_Centroid_sum_x,K*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_Centroid_sum_y,d_Centroid_sum_y,K*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_Cluster_Size,d_Cluster_Size,K*sizeof(int),hipMemcpyDeviceToHost);

        hipMemset(d_Cluster_Size,0,K*sizeof(int));
        for(int i=0; i < K; i++){
            h_Centroid_Coord_X[i]=h_Centroid_sum_x[i]/h_Cluster_Size[i];
            h_Centroid_Coord_Y[i]=h_Centroid_sum_y[i]/h_Cluster_Size[i];
        }
        for(int i=0; i < K; i++){
            //printf("C %d: (%f, %f)\n",i,h_Centroid_Coord_X[i],h_Centroid_Coord_Y[i]);
        }

        //Stop time for iteration
        clock_t end_iter = clock();
        float seconds_iter = (float)(end_iter - start_iter) / CLOCKS_PER_SEC/1000;
        //printf("Time for iter: %f\n", seconds_iter);

        //Compare the centroids for stop the clustering
        hipMemcpy(d_Centroid_Coord_X,h_Centroid_Coord_X,K*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(d_Centroid_Coord_Y,h_Centroid_Coord_Y,K*sizeof(float),hipMemcpyHostToDevice);

        cur_iter+=1;
    }

    hipMemcpy(h_Cluster_Membership,d_Cluster_Membership,N*sizeof(int),hipMemcpyDeviceToHost);

    clock_t end = clock();
    float milliseconds = (float)(end - start) / CLOCKS_PER_SEC * 1000;
    printf("Time for clustering: %f milliseconds \n", milliseconds);
    printf("Time for average iteration: %f milliseconds\n", milliseconds / MAX_ITER);
    FILE *res;

    res = fopen("/content/drive/My Drive/Parallel/K-means/Cuda/2D_data_3_results.csv", "w+");
    for(int i=0;i<N;i++){
        fprintf(res,"%d\n", h_Cluster_Membership[i]);
    }

    hipFree(d_Point_Coord_x);
    hipFree(d_Point_Coord_y);
    hipFree(d_Cluster_Membership);
    hipFree(d_Centroid_Coord_X);
    hipFree(d_Centroid_Coord_Y);
    hipFree(d_Cluster_Size);

    free(h_Centroid_Coord_X);
    free(h_Centroid_Coord_Y);
    free(h_Point_Coord_x);
    free(h_Point_Coord_y);
    free(h_Cluster_Membership);
    free(h_Cluster_Size);

    return 0;
}